#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <torch/extension.h>
#include <math.h>
#include <stdio.h>
#include "neuron_def.h"
__forceinline__  __device__ float grad_atan(const float & alpha, const float & x)
{
  const float M_PI_2__alpha__x = (float) M_PI_2 * alpha * x;
  return alpha / 2.0f / (1.0f + M_PI_2__alpha__x * M_PI_2__alpha__x);
}

__forceinline__  __device__ float grad_sigmoid(const float & alpha, const float & x)
{
  const float sigmoid_ax = 1.0f / (1.0f + expf(- alpha * x));
  return (1.0f - sigmoid_ax) * sigmoid_ax * alpha;
}

typedef float (*grad_surrogate_function) (const float &, const float &);

__device__ const grad_surrogate_function grad_surrogate_function_pointer[2] = { 
    grad_atan, 
    grad_sigmoid
    };


__forceinline__  __device__ half grad_atan_half(const half & alpha, const half & x)
{
  #if __CUDACC_VER_MAJOR__ >= 11
  const half M_PI_2__alpha__x = __hmul(__hmul(__double2half(M_PI_2), alpha), x);
  #else
  const half M_PI_2__alpha__x = __hmul(__hmul(__float2half((float) M_PI_2), alpha), x);
  #endif
  return __hdiv(__hdiv(alpha, __float2half(2.0f)), __hfma(M_PI_2__alpha__x, M_PI_2__alpha__x, __float2half(1.0f)));
}

__forceinline__  __device__ half grad_sigmoid_half(const half & alpha, const half & x)
{
  const half sigmoid_ax = __hdiv(__float2half(1.0f), __hadd(hexp(__hneg(__hmul(alpha, x))), __float2half(1.0f)));
  return __hmul(__hmul(__hsub(__float2half(1.0f), sigmoid_ax), sigmoid_ax), alpha);
}

typedef half (*grad_surrogate_function_half) (const half &, const half &);

__device__ const grad_surrogate_function_half grad_surrogate_function_pointer_half[2] = { 
    grad_atan_half, 
    grad_sigmoid_half
    };

//LIF hard reset----------------------------------------------------
__global__ void LIF_hard_reset_forward_cuda_kernel(
    const float* __restrict__ x, const float* __restrict__ v, float* __restrict__ spike, float* __restrict__ v_next, 
    const float v_th, const float v_reset, const int size,
    const float reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const float h = v[index] + reciprocal_tau * (x[index] - v[index] + v_reset);
    if (h >= v_th)
    {
      spike[index] = 1.0f;
      v_next[index] = v_reset;
    }
    else
    {
      spike[index] = 0.0f;
      v_next[index] = h;
    }
  }
}

__global__ void LIF_hard_reset_forward_cuda_kernel_half(
  const at::Half* __restrict__ x, const at::Half* __restrict__ v, at::Half* __restrict__ spike, at::Half* __restrict__ v_next, 
  const half v_th, const half v_reset, const int size,
  const half reciprocal_tau)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const half h = __hfma(reciprocal_tau, __hadd(__hsub(x[index], v[index]), v_reset), v[index]);
  if (__hgeu(h, v_th))
  {
    spike[index] = __float2half(1.0f);
    v_next[index] = v_reset;
  }
  else
  {
    spike[index] = __float2half(0.0f);
    v_next[index] = h;
  }
}
}

//LIF hard reset detach x----------------------------------------------------

__global__ void LIF_detach_x_hard_reset_forward_cuda_kernel(
  const float* __restrict__ x, const float* __restrict__ v, float* __restrict__ spike, float* __restrict__ v_next, 
  const float v_th, const float v_reset, const int size,
  const float reciprocal_tau)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const float h = v[index] + x[index] + reciprocal_tau * (v_reset - v[index]);
  if (h >= v_th)
  {
    spike[index] = 1.0f;
    v_next[index] = v_reset;
  }
  else
  {
    spike[index] = 0.0f;
    v_next[index] = h;
  }
}
}

__global__ void LIF_detach_x_hard_reset_forward_cuda_kernel_half(
const at::Half* __restrict__ x, const at::Half* __restrict__ v, at::Half* __restrict__ spike, at::Half* __restrict__ v_next, 
const half v_th, const half v_reset, const int size,
const half reciprocal_tau)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
const half h = __hfma(reciprocal_tau, __hsub(v_reset, v[index]), __hadd((half) v[index], (half) x[index]));
if (__hgeu(h, v_th))
{
  spike[index] = __float2half(1.0f);
  v_next[index] = v_reset;
}
else
{
  spike[index] = __float2half(0.0f);
  v_next[index] = h;
}
}
}

std::vector<at::Tensor> LIF_hard_reset_forward(torch::Tensor & x, torch::Tensor & v, const float & v_th, const float & v_reset, 
  const float & reciprocal_tau, const bool & detach_x)
{   
  CHECK_TENSOR(x);
  CHECK_TENSOR(v);
  auto spike = torch::zeros_like(v.data());
  auto v_next = torch::zeros_like(v.data());
  CHECK_TENSOR(spike);
  CHECK_TENSOR(v_next);
  const int size = x.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(x.get_device()));
  if (x.scalar_type() == c10::ScalarType::Float)
  {
    if (detach_x)
    {
      LIF_detach_x_hard_reset_forward_cuda_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), v.data_ptr<float>(), spike.data_ptr<float>(), v_next.data_ptr<float>(), 
        v_th, v_reset, size, reciprocal_tau);
    }
    else
    {
      LIF_hard_reset_forward_cuda_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), v.data_ptr<float>(), spike.data_ptr<float>(), v_next.data_ptr<float>(), 
        v_th, v_reset, size, reciprocal_tau);
    }

  }
  else if (x.scalar_type() == c10::ScalarType::Half)
  {
    if (detach_x)
    {
      LIF_detach_x_hard_reset_forward_cuda_kernel_half<<<blocks, threads>>>(
        x.data_ptr<at::Half>(), v.data_ptr<at::Half>(), spike.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), 
        __float2half(v_th), __float2half(v_reset), size, __float2half(reciprocal_tau));
    }
    else
    {
      LIF_hard_reset_forward_cuda_kernel_half<<<blocks, threads>>>(
        x.data_ptr<at::Half>(), v.data_ptr<at::Half>(), spike.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), 
        __float2half(v_th), __float2half(v_reset), size, __float2half(reciprocal_tau));
    }
  }
  return {spike, v_next};
}

__global__ void LIF_hard_reset_forward_with_grad_cuda_kernel(
  const float* __restrict__ x, const float* __restrict__ v, float* __restrict__ spike, float* __restrict__ v_next,
  float* __restrict__ grad_s_to_h, float* __restrict__ grad_v_to_h,
  const float v_th, const float v_reset, const int size,
  const float alpha, const bool detach_reset, const int grad_surrogate_function_index,
  const float reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const float h = v[index] + reciprocal_tau * (x[index] - v[index] + v_reset);
    if (h >= v_th)
    {
      spike[index] = 1.0f;
      v_next[index] = v_reset;
    }
    else
    {
      spike[index] = 0.0f;
      v_next[index] = h;
    }
    grad_s_to_h[index] = grad_surrogate_function_pointer[grad_surrogate_function_index](alpha, h - v_th);
    grad_v_to_h[index] = 1.0f - spike[index] + (v_reset - h) * grad_s_to_h[index] * (1.0f - (float) detach_reset);
  }
}

__global__ void LIF_hard_reset_forward_with_grad_cuda_kernel_half(
  const at::Half* __restrict__ x, const at::Half* __restrict__ v, at::Half* __restrict__ spike, at::Half* __restrict__ v_next,
  at::Half* __restrict__ grad_s_to_h, at::Half* __restrict__ grad_v_to_h,
  const half v_th, const half v_reset, const int size,
  const half alpha, const bool detach_reset, const int grad_surrogate_function_index,
  const half reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const half h = __hfma(reciprocal_tau, __hadd(__hsub(x[index], v[index]), v_reset), v[index]);
    if (__hgeu(h, v_th))
    {
      spike[index] = __float2half(1.0f);
      v_next[index] = v_reset;
    }
    else
    {
      spike[index] = __float2half(0.0f);
      v_next[index] = h;
    }

    grad_s_to_h[index] = grad_surrogate_function_pointer_half[grad_surrogate_function_index](alpha, __hsub(h, v_th));
    grad_v_to_h[index] = __hfma(__hmul(__hsub(v_reset, h), grad_s_to_h[index]), __float2half(1.0f - (float) detach_reset), __hsub(__float2half(1.0f), spike[index]));
  }
}

// detach x---------

__global__ void LIF_detach_x_hard_reset_forward_with_grad_cuda_kernel(
  const float* __restrict__ x, const float* __restrict__ v, float* __restrict__ spike, float* __restrict__ v_next,
  float* __restrict__ grad_s_to_h, float* __restrict__ grad_v_to_h,
  const float v_th, const float v_reset, const int size,
  const float alpha, const bool detach_reset, const int grad_surrogate_function_index,
  const float reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const float h = v[index] + x[index] + reciprocal_tau * (v_reset - v[index]);
    if (h >= v_th)
    {
      spike[index] = 1.0f;
      v_next[index] = v_reset;
    }
    else
    {
      spike[index] = 0.0f;
      v_next[index] = h;
    }
    grad_s_to_h[index] = grad_surrogate_function_pointer[grad_surrogate_function_index](alpha, h - v_th);
    grad_v_to_h[index] = 1.0f - spike[index] + (v_reset - h) * grad_s_to_h[index] * (1.0f - (float) detach_reset);
  }
}

__global__ void LIF_detach_x_hard_reset_forward_with_grad_cuda_kernel_half(
  const at::Half* __restrict__ x, const at::Half* __restrict__ v, at::Half* __restrict__ spike, at::Half* __restrict__ v_next,
  at::Half* __restrict__ grad_s_to_h, at::Half* __restrict__ grad_v_to_h,
  const half v_th, const half v_reset, const int size,
  const half alpha, const bool detach_reset, const int grad_surrogate_function_index,
  const half reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const half h = __hfma(reciprocal_tau, __hsub(v_reset, v[index]), __hadd((half) v[index], (half) x[index]));
    if (__hgeu(h, v_th))
    {
      spike[index] = __float2half(1.0f);
      v_next[index] = v_reset;
    }
    else
    {
      spike[index] = __float2half(0.0f);
      v_next[index] = h;
    }

    grad_s_to_h[index] = grad_surrogate_function_pointer_half[grad_surrogate_function_index](alpha, __hsub(h, v_th));
    grad_v_to_h[index] = __hfma(__hmul(__hsub(v_reset, h), grad_s_to_h[index]), __float2half(1.0f - (float) detach_reset), __hsub(__float2half(1.0f), spike[index]));
  }
}

std::vector<at::Tensor> LIF_hard_reset_forward_with_grad(torch::Tensor & x, torch::Tensor & v, const float & v_th, const float & v_reset,
  const float & alpha, const bool & detach_reset, const int & grad_surrogate_function_index,
  const float & reciprocal_tau, const bool & detach_x)
{   
  CHECK_TENSOR(x);
  CHECK_TENSOR(v);

  auto spike = torch::zeros_like(v.data());
  auto v_next = spike.data().clone();
  auto grad_s_to_h = spike.data().clone();
  auto grad_v_to_h = spike.data().clone();

  CHECK_TENSOR(spike);
  CHECK_TENSOR(v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  const int size = x.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(x.get_device()));
  if (x.scalar_type() == c10::ScalarType::Float)
  {
    if (detach_x)
    {
      LIF_detach_x_hard_reset_forward_with_grad_cuda_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), v.data_ptr<float>(), spike.data_ptr<float>(), v_next.data_ptr<float>(), 
        grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(), 
        v_th, v_reset, size, 
        alpha, detach_reset, grad_surrogate_function_index,
        reciprocal_tau);
    }
    else
    {
      LIF_hard_reset_forward_with_grad_cuda_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), v.data_ptr<float>(), spike.data_ptr<float>(), v_next.data_ptr<float>(), 
        grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(), 
        v_th, v_reset, size, 
        alpha, detach_reset, grad_surrogate_function_index,
        reciprocal_tau);
    }

  }
  else if (x.scalar_type() == c10::ScalarType::Half)
  {
    if (detach_x)
    {
      LIF_detach_x_hard_reset_forward_with_grad_cuda_kernel_half<<<blocks, threads>>>(
        x.data_ptr<at::Half>(), v.data_ptr<at::Half>(), spike.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), 
        grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(), 
        __float2half(v_th), __float2half(v_reset), size, 
        __float2half(alpha), detach_reset, grad_surrogate_function_index,
        __float2half(reciprocal_tau));
    }
    else
    {
      LIF_hard_reset_forward_with_grad_cuda_kernel_half<<<blocks, threads>>>(
        x.data_ptr<at::Half>(), v.data_ptr<at::Half>(), spike.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), 
        grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(), 
        __float2half(v_th), __float2half(v_reset), size, 
        __float2half(alpha), detach_reset, grad_surrogate_function_index,
        __float2half(reciprocal_tau));
    }

  }
  return {spike, v_next, grad_s_to_h, grad_v_to_h};

}

//IF hard reset----------------------------------------------------
__global__ void IF_hard_reset_forward_cuda_kernel(
  const float* __restrict__ x, const float* __restrict__ v, float* __restrict__ spike, float* __restrict__ v_next, 
  const float v_th, const float v_reset, const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const float h = v[index] + x[index];
  if (h >= v_th)
  {
    spike[index] = 1.0f;
    v_next[index] = v_reset;
  }
  else
  {
    spike[index] = 0.0f;
    v_next[index] = h;
  }
}
}

__global__ void IF_hard_reset_forward_cuda_kernel_half(
  const at::Half* __restrict__ x, const at::Half* __restrict__ v, at::Half* __restrict__ spike, at::Half* __restrict__ v_next, 
  const half v_th, const half v_reset, const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const half h = __hadd((half) v[index], (half) x[index]);
  if (__hgeu(h, v_th))
  {
    spike[index] = __float2half(1.0f);
    v_next[index] = v_reset;
  }
  else
  {
    spike[index] = __float2half(0.0f);
    v_next[index] = h;
  }
}
}

std::vector<at::Tensor> IF_hard_reset_forward(torch::Tensor & x, torch::Tensor & v, const float & v_th, const float & v_reset)
{   
    CHECK_TENSOR(x);
    CHECK_TENSOR(v);
    auto spike = torch::zeros_like(v.data());
    auto v_next = torch::zeros_like(v.data());
    CHECK_TENSOR(spike);
    CHECK_TENSOR(v_next);
    const int size = x.numel();
    const int threads = THREADS;
    const int blocks = (size + threads - 1) / threads;
    CHECK_CUDA_OPERATION(hipSetDevice(x.get_device()));
    if (x.scalar_type() == c10::ScalarType::Float)
    {
      IF_hard_reset_forward_cuda_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), v.data_ptr<float>(), spike.data_ptr<float>(), v_next.data_ptr<float>(),
        v_th, v_reset, size);
    }
    else if (x.scalar_type() == c10::ScalarType::Half)
    {
      IF_hard_reset_forward_cuda_kernel_half<<<blocks, threads>>>(
        x.data_ptr<at::Half>(), v.data_ptr<at::Half>(), spike.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(),
        __float2half(v_th), __float2half(v_reset), size);
    }

    return {spike, v_next};
}

__global__ void IF_hard_reset_forward_with_grad_cuda_kernel(
const float* __restrict__ x, const float* __restrict__ v, float* __restrict__ spike, float* __restrict__ v_next,
float* __restrict__ grad_s_to_h, float* __restrict__ grad_v_to_h,
const float v_th, const float v_reset, const int size,
const float alpha, const bool detach_reset, const int grad_surrogate_function_index)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const float h = v[index] + x[index];
  if (h >= v_th)
  {
    spike[index] = 1.0f;
    v_next[index] = v_reset;
  }
  else
  {
    spike[index] = 0.0f;
    v_next[index] = h;
  }
  grad_s_to_h[index] = grad_surrogate_function_pointer[grad_surrogate_function_index](alpha, h - v_th);
  grad_v_to_h[index] = 1.0f - spike[index] + (v_reset - h) * grad_s_to_h[index] * (1.0f - (float) detach_reset);
}
}

__global__ void IF_hard_reset_forward_with_grad_cuda_kernel_half(
  const at::Half* __restrict__ x, const at::Half* __restrict__ v, at::Half* __restrict__ spike, at::Half* __restrict__ v_next,
  at::Half* __restrict__ grad_s_to_h, at::Half* __restrict__ grad_v_to_h,
  const half v_th, const half v_reset, const int size,
  const half alpha, const bool detach_reset, const int grad_surrogate_function_index)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const half h = __hadd((half) v[index], (half) x[index]);
    if (__hgeu(h, v_th))
    {
      spike[index] = __float2half(1.0f);
      v_next[index] = v_reset;
    }
    else
    {
      spike[index] = __float2half(0.0f);
      v_next[index] = h;
    }
    grad_s_to_h[index] = grad_surrogate_function_pointer_half[grad_surrogate_function_index](alpha, __hsub(h, v_th));
    grad_v_to_h[index] = __hfma(__hmul(__hsub(v_reset, h), grad_s_to_h[index]), __float2half(1.0f - (float) detach_reset), __hsub(__float2half(1.0f), spike[index]));
  }
}

std::vector<at::Tensor> IF_hard_reset_forward_with_grad(torch::Tensor & x, torch::Tensor & v, const float & v_th, const float & v_reset,
  const float & alpha, const bool & detach_reset, const int & grad_surrogate_function_index)
{   
  CHECK_TENSOR(x);
  CHECK_TENSOR(v);

  auto spike = torch::zeros_like(v.data());
  auto v_next = spike.data().clone();
  auto grad_s_to_h = spike.data().clone();
  auto grad_v_to_h = spike.data().clone();

  CHECK_TENSOR(spike);
  CHECK_TENSOR(v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  const int size = x.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(x.get_device()));
  if (x.scalar_type() == c10::ScalarType::Float)
  {
    IF_hard_reset_forward_with_grad_cuda_kernel<<<blocks, threads>>>(
      x.data_ptr<float>(), v.data_ptr<float>(), spike.data_ptr<float>(), v_next.data_ptr<float>(), 
      grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(), 
      v_th, v_reset, size, 
      alpha, detach_reset, grad_surrogate_function_index);
  }
  else if (x.scalar_type() == c10::ScalarType::Half)
  {
    IF_hard_reset_forward_with_grad_cuda_kernel_half<<<blocks, threads>>>(
      x.data_ptr<at::Half>(), v.data_ptr<at::Half>(), spike.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), 
      grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(), 
      __float2half(v_th), __float2half(v_reset), size, 
      __float2half(alpha), detach_reset, grad_surrogate_function_index);
  }

  return {spike, v_next, grad_s_to_h, grad_v_to_h};
}

//LIF hard reset fptt----------------------------------------------------
__global__ void LIF_hard_reset_fptt_cuda_kernel(
  const float* __restrict__ x_seq, float* __restrict__ spike_seq, float* __restrict__ v_next, 
  const float v_th, const float v_reset, const int neuron_num, const int size,
  const float reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const float h = v_next[index] + reciprocal_tau * (x_seq[mem_index] - v_next[index] + v_reset);
      if (h >= v_th)
      {
        spike_seq[mem_index] = 1.0f;
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = 0.0f;
        v_next[index] = h;
      }
    }
    
  }
}

__global__ void LIF_hard_reset_fptt_cuda_kernel_half(
  const at::Half* __restrict__ x_seq, at::Half* __restrict__ spike_seq, at::Half* __restrict__ v_next, 
  const half v_th, const half v_reset, const int neuron_num, const int size,
  const half reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const half h = __hfma(reciprocal_tau, __hadd(__hsub(x_seq[mem_index], v_next[index]), v_reset), v_next[index]);
      if (__hgeu(h, v_th))
      {
        spike_seq[mem_index] = __float2half(1.0f);
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = __float2half(0.0f);
        v_next[index] = h;
      }
    }
    
  }
}

//detach x------

__global__ void LIF_detach_x_hard_reset_fptt_cuda_kernel(
  const float* __restrict__ x_seq, float* __restrict__ spike_seq, float* __restrict__ v_next, 
  const float v_th, const float v_reset, const int neuron_num, const int size,
  const float reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const float h = v_next[index] + x_seq[mem_index] + reciprocal_tau * (v_reset - v_next[index]);
      if (h >= v_th)
      {
        spike_seq[mem_index] = 1.0f;
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = 0.0f;
        v_next[index] = h;
      }
    }
    
  }
}

__global__ void LIF_detach_x_hard_reset_fptt_cuda_kernel_half(
  const at::Half* __restrict__ x_seq, at::Half* __restrict__ spike_seq, at::Half* __restrict__ v_next, 
  const half v_th, const half v_reset, const int neuron_num, const int size,
  const half reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const half h = __hfma(reciprocal_tau, __hsub(v_reset, v_next[index]), __hadd((half) v_next[index], (half) x_seq[mem_index]));

      if (__hgeu(h, v_th))
      {
        spike_seq[mem_index] = __float2half(1.0f);
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = __float2half(0.0f);
        v_next[index] = h;
      }
    }
    
  }
}

std::vector<at::Tensor> LIF_hard_reset_fptt(torch::Tensor & x_seq, torch::Tensor & v, const float & v_th, const float & v_reset, 
  const float & reciprocal_tau, const bool & detach_x)
{
  CHECK_TENSOR(x_seq);
  CHECK_TENSOR(v);
  auto spike_seq = torch::zeros_like(x_seq.data());
  auto v_next = v.data().clone();
  CHECK_TENSOR(spike_seq);
  CHECK_TENSOR(v_next);
  const int seq_len = x_seq.size(0);
  const int size = x_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(x_seq.get_device()));
  if (x_seq.scalar_type() == c10::ScalarType::Float)
  {
    if (detach_x)
    {
      LIF_detach_x_hard_reset_fptt_cuda_kernel<<<blocks, threads>>>(
        x_seq.data_ptr<float>(), spike_seq.data_ptr<float>(), v_next.data_ptr<float>(),
        v_th, v_reset, neuron_num, size, reciprocal_tau);
    }
    else
    {
      LIF_hard_reset_fptt_cuda_kernel<<<blocks, threads>>>(
        x_seq.data_ptr<float>(), spike_seq.data_ptr<float>(), v_next.data_ptr<float>(),
        v_th, v_reset, neuron_num, size, reciprocal_tau);
    }

  }
  else if (x_seq.scalar_type() == c10::ScalarType::Half)
  {
    if (detach_x)
    {
      LIF_detach_x_hard_reset_fptt_cuda_kernel_half<<<blocks, threads>>>(
        x_seq.data_ptr<at::Half>(), spike_seq.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(),
        __float2half(v_th), __float2half(v_reset), neuron_num, size, __float2half(reciprocal_tau));
    }
    else
    {
      LIF_hard_reset_fptt_cuda_kernel_half<<<blocks, threads>>>(
        x_seq.data_ptr<at::Half>(), spike_seq.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(),
        __float2half(v_th), __float2half(v_reset), neuron_num, size, __float2half(reciprocal_tau));
    }

  }

  return {spike_seq, v_next};
}

__global__ void LIF_hard_reset_fptt_with_grad_cuda_kernel(
  const float* __restrict__ x_seq, float* __restrict__ spike_seq, float* __restrict__ v_next, float* __restrict__ grad_s_to_h, float* __restrict__ grad_v_to_h, 
  const float v_th, const float v_reset, const int neuron_num, const int size,
  const float alpha, const bool detach_reset, const int grad_surrogate_function_index,
  const float reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const float h = v_next[index] + reciprocal_tau * (x_seq[mem_index] - v_next[index] + v_reset);
      if (h >= v_th)
      {
        spike_seq[mem_index] = 1.0f;
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = 0.0f;
        v_next[index] = h;
      }
      grad_s_to_h[mem_index] = grad_surrogate_function_pointer[grad_surrogate_function_index](alpha, h - v_th);
      grad_v_to_h[mem_index] = 1.0f - spike_seq[mem_index] + (v_reset - h) * grad_s_to_h[mem_index] * (1.0f - (float) detach_reset);
    }
    
  }
}

__global__ void LIF_hard_reset_fptt_with_grad_cuda_kernel_half(
  const at::Half* __restrict__ x_seq, at::Half* __restrict__ spike_seq, at::Half* __restrict__ v_next, at::Half* __restrict__ grad_s_to_h, at::Half* __restrict__ grad_v_to_h, 
  const half v_th, const half v_reset, const int neuron_num, const int size,
  const half alpha, const bool detach_reset, const int grad_surrogate_function_index,
  const half reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const half h = __hfma(reciprocal_tau, __hadd(__hsub(x_seq[mem_index], v_next[index]), v_reset), v_next[index]);
      if (__hgeu(h, v_th))
      {
        spike_seq[mem_index] = __float2half(1.0f);
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = __float2half(0.0f);
        v_next[index] = h;
      }
      grad_s_to_h[mem_index] = grad_surrogate_function_pointer_half[grad_surrogate_function_index](alpha, __hsub(h, v_th));
      grad_v_to_h[mem_index] = __hfma(__hmul(__hsub(v_reset, h), grad_s_to_h[mem_index]), __float2half(1.0f - (float) detach_reset), __hsub(__float2half(1.0f), spike_seq[mem_index]));
    }
    
  }
}

//detach x--------
__global__ void LIF_detach_x_hard_reset_fptt_with_grad_cuda_kernel(
  const float* __restrict__ x_seq, float* __restrict__ spike_seq, float* __restrict__ v_next, float* __restrict__ grad_s_to_h, float* __restrict__ grad_v_to_h, 
  const float v_th, const float v_reset, const int neuron_num, const int size,
  const float alpha, const bool detach_reset, const int grad_surrogate_function_index,
  const float reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const float h = v_next[index] + x_seq[mem_index] + reciprocal_tau * (v_reset - v_next[index]);

      if (h >= v_th)
      {
        spike_seq[mem_index] = 1.0f;
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = 0.0f;
        v_next[index] = h;
      }
      grad_s_to_h[mem_index] = grad_surrogate_function_pointer[grad_surrogate_function_index](alpha, h - v_th);
      grad_v_to_h[mem_index] = 1.0f - spike_seq[mem_index] + (v_reset - h) * grad_s_to_h[mem_index] * (1.0f - (float) detach_reset);
    }
    
  }
}

__global__ void LIF_detach_x_hard_reset_fptt_with_grad_cuda_kernel_half(
  const at::Half* __restrict__ x_seq, at::Half* __restrict__ spike_seq, at::Half* __restrict__ v_next, at::Half* __restrict__ grad_s_to_h, at::Half* __restrict__ grad_v_to_h, 
  const half v_th, const half v_reset, const int neuron_num, const int size,
  const half alpha, const bool detach_reset, const int grad_surrogate_function_index,
  const half reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const half h = __hfma(reciprocal_tau, __hsub(v_reset, v_next[index]), __hadd((half) v_next[index], (half) x_seq[mem_index]));

      if (__hgeu(h, v_th))
      {
        spike_seq[mem_index] = __float2half(1.0f);
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = __float2half(0.0f);
        v_next[index] = h;
      }
      grad_s_to_h[mem_index] = grad_surrogate_function_pointer_half[grad_surrogate_function_index](alpha, __hsub(h, v_th));
      grad_v_to_h[mem_index] = __hfma(__hmul(__hsub(v_reset, h), grad_s_to_h[mem_index]), __float2half(1.0f - (float) detach_reset), __hsub(__float2half(1.0f), spike_seq[mem_index]));
    }
    
  }
}

std::vector<at::Tensor> LIF_hard_reset_fptt_with_grad(torch::Tensor & x_seq, torch::Tensor & v, const float & v_th, const float & v_reset, 
  const float & alpha, const bool & detach_reset, const int & grad_surrogate_function_index,
  const float & reciprocal_tau, const bool & detach_x)
{
  CHECK_TENSOR(x_seq);
  CHECK_TENSOR(v);
  auto spike_seq = torch::zeros_like(x_seq.data());
  auto v_next = v.data().clone();
  auto grad_s_to_h = spike_seq.data().clone();
  auto grad_v_to_h = spike_seq.data().clone();
  CHECK_TENSOR(spike_seq);
  CHECK_TENSOR(v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  const int seq_len = x_seq.size(0);
  const int size = x_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(x_seq.get_device()));
  if (x_seq.scalar_type() == c10::ScalarType::Float)
  {
    if (detach_x)
    {
      LIF_detach_x_hard_reset_fptt_with_grad_cuda_kernel<<<blocks, threads>>>(
        x_seq.data_ptr<float>(), spike_seq.data_ptr<float>(), v_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
        v_th, v_reset, neuron_num, size, 
        alpha, detach_reset, grad_surrogate_function_index,
        reciprocal_tau);
    }
    else
    {
      LIF_hard_reset_fptt_with_grad_cuda_kernel<<<blocks, threads>>>(
        x_seq.data_ptr<float>(), spike_seq.data_ptr<float>(), v_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
        v_th, v_reset, neuron_num, size, 
        alpha, detach_reset, grad_surrogate_function_index,
        reciprocal_tau);
    }

  }
  else if (x_seq.scalar_type() == c10::ScalarType::Half)
  {
    if (detach_x)
    {
      LIF_detach_x_hard_reset_fptt_with_grad_cuda_kernel_half<<<blocks, threads>>>(
        x_seq.data_ptr<at::Half>(), spike_seq.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
        __float2half(v_th), __float2half(v_reset), neuron_num, size, 
        __float2half(alpha), detach_reset, grad_surrogate_function_index,
        __float2half(reciprocal_tau));
    }
    else
    {
      LIF_hard_reset_fptt_with_grad_cuda_kernel_half<<<blocks, threads>>>(
        x_seq.data_ptr<at::Half>(), spike_seq.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
        __float2half(v_th), __float2half(v_reset), neuron_num, size, 
        __float2half(alpha), detach_reset, grad_surrogate_function_index,
        __float2half(reciprocal_tau));
    }

  }

  return {spike_seq, v_next, grad_s_to_h, grad_v_to_h};
}
//IF hard reset fptt----------------------------------------------------
__global__ void IF_hard_reset_fptt_cuda_kernel(
  const float* __restrict__ x_seq, float* __restrict__ spike_seq, float* __restrict__ v_next, 
  const float v_th, const float v_reset, const int neuron_num, const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const float h = v_next[index] + x_seq[mem_index];
      if (h >= v_th)
      {
        spike_seq[mem_index] = 1.0f;
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = 0.0f;
        v_next[index] = h;
      }
    }
    
  }
}

__global__ void IF_hard_reset_fptt_cuda_kernel_half(
  const at::Half* __restrict__ x_seq, at::Half* __restrict__ spike_seq, at::Half* __restrict__ v_next, 
  const half v_th, const half v_reset, const int neuron_num, const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const half h = __hadd((half) v_next[index], (half) x_seq[mem_index]);
      if (__hgeu(h, v_th))
      {
        spike_seq[mem_index] = __float2half(1.0f);
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = __float2half(0.0f);
        v_next[index] = h;
      }
    }
    
  }
}

std::vector<at::Tensor> IF_hard_reset_fptt(torch::Tensor & x_seq, torch::Tensor & v, const float & v_th, const float & v_reset)
{
    CHECK_TENSOR(x_seq);
    CHECK_TENSOR(v);
    auto spike_seq = torch::zeros_like(x_seq.data());
    auto v_next = v.data().clone();
    CHECK_TENSOR(spike_seq);
    CHECK_TENSOR(v_next);
    const int seq_len = x_seq.size(0);
    const int size = x_seq.numel();
    const int threads = THREADS;
    const int neuron_num = size / seq_len;
    const int blocks = (neuron_num + threads - 1) / threads;
    CHECK_CUDA_OPERATION(hipSetDevice(x_seq.get_device()));
    if (x_seq.scalar_type() == c10::ScalarType::Float)
    {
      IF_hard_reset_fptt_cuda_kernel<<<blocks, threads>>>(
        x_seq.data_ptr<float>(), spike_seq.data_ptr<float>(), v_next.data_ptr<float>(),
        v_th, v_reset, neuron_num, size);
    }
    else if (x_seq.scalar_type() == c10::ScalarType::Half)
    {
      IF_hard_reset_fptt_cuda_kernel_half<<<blocks, threads>>>(
        x_seq.data_ptr<at::Half>(), spike_seq.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(),
        __float2half(v_th), __float2half(v_reset), neuron_num, size);
    }

    return {spike_seq, v_next};
}

__global__ void IF_hard_reset_fptt_with_grad_cuda_kernel(
  const float* __restrict__ x_seq, float* __restrict__ spike_seq, float* __restrict__ v_next, float* __restrict__ grad_s_to_h, float* __restrict__ grad_v_to_h, 
  const float v_th, const float v_reset, const int neuron_num, const int size,
  const float alpha, const bool detach_reset, const int grad_surrogate_function_index)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const float h = v_next[index] + x_seq[mem_index];
      if (h >= v_th)
      {
        spike_seq[mem_index] = 1.0f;
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = 0.0f;
        v_next[index] = h;
      }
      grad_s_to_h[mem_index] = grad_surrogate_function_pointer[grad_surrogate_function_index](alpha, h - v_th);
      grad_v_to_h[mem_index] = 1.0f - spike_seq[mem_index] + (v_reset - h) * grad_s_to_h[mem_index] * (1.0f - (float) detach_reset);
    }
    
  }
}

__global__ void IF_hard_reset_fptt_with_grad_cuda_kernel_half(
  const at::Half* __restrict__ x_seq, at::Half* __restrict__ spike_seq, at::Half* __restrict__ v_next, at::Half* __restrict__ grad_s_to_h, at::Half* __restrict__ grad_v_to_h, 
  const half v_th, const half v_reset, const int neuron_num, const int size,
  const half alpha, const bool detach_reset, const int grad_surrogate_function_index)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const half h = __hadd((half) v_next[index], (half) x_seq[mem_index]);
      if (__hgeu(h, v_th))
      {
        spike_seq[mem_index] = __float2half(1.0f);
        v_next[index] = v_reset;
      }
      else
      {
        spike_seq[mem_index] = __float2half(0.0f);
        v_next[index] = h;
      }
      grad_s_to_h[mem_index] = grad_surrogate_function_pointer_half[grad_surrogate_function_index](alpha, __hsub(h, v_th));
      grad_v_to_h[mem_index] = __hfma(__hmul(__hsub(v_reset, h), grad_s_to_h[mem_index]), __float2half(1.0f - (float) detach_reset), __hsub(__float2half(1.0f), spike_seq[mem_index]));
    }
    
  }
}

std::vector<at::Tensor> IF_hard_reset_fptt_with_grad(torch::Tensor & x_seq, torch::Tensor & v, const float & v_th, const float & v_reset, 
  const float & alpha, const bool & detach_reset, const int & grad_surrogate_function_index)
{
  CHECK_TENSOR(x_seq);
  CHECK_TENSOR(v);
  auto spike_seq = torch::zeros_like(x_seq.data());
  auto v_next = v.data().clone();
  auto grad_s_to_h = spike_seq.data().clone();
  auto grad_v_to_h = spike_seq.data().clone();
  CHECK_TENSOR(spike_seq);
  CHECK_TENSOR(v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  const int seq_len = x_seq.size(0);
  const int size = x_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(x_seq.get_device()));
  if (x_seq.scalar_type() == c10::ScalarType::Float)
  {
    IF_hard_reset_fptt_with_grad_cuda_kernel<<<blocks, threads>>>(
      x_seq.data_ptr<float>(), spike_seq.data_ptr<float>(), v_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
      v_th, v_reset, neuron_num, size, 
      alpha, detach_reset, grad_surrogate_function_index);
  }
  else if (x_seq.scalar_type() == c10::ScalarType::Half)
  {
    IF_hard_reset_fptt_with_grad_cuda_kernel_half<<<blocks, threads>>>(
      x_seq.data_ptr<at::Half>(), spike_seq.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
      __float2half(v_th), __float2half(v_reset), neuron_num, size, 
      __float2half(alpha), detach_reset, grad_surrogate_function_index);
  }

  return {spike_seq, v_next, grad_s_to_h, grad_v_to_h};
}


//OneSpikeIF hard reset----------------------------------------------------
__global__ void OneSpikeIF_hard_reset_forward_cuda_kernel(
  const float* __restrict__ x, const float* __restrict__ v, const float* __restrict__ fire_mask, float* __restrict__ spike, float* __restrict__ v_next, float* __restrict__ fire_mask_next, 
  const float v_th, const float v_reset, const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const float h = v[index] + x[index];
  if (h >= v_th)
  {
    spike[index] = 1.0f - fire_mask[index];
    v_next[index] = h + (v_reset - h) * spike[index];
    // v_next[index] = v_reset * spike[index] + (1.0f - spike[index]) * h;
    fire_mask_next[index] = 1.0f;
  }
  else
  {
    spike[index] = 0.0f;
    v_next[index] = h;
  }
}
}

__global__ void OneSpikeIF_hard_reset_forward_cuda_kernel_half(
  const at::Half* __restrict__ x, const at::Half* __restrict__ v, const at::Half* __restrict__ fire_mask, at::Half* __restrict__ spike, at::Half* __restrict__ v_next, at::Half* __restrict__ fire_mask_next, 
  const half v_th, const half v_reset, const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const half h = __hadd((half) v[index], (half) x[index]);
  if (__hgeu(h, v_th))
  {
    spike[index] = __hsub(__float2half(1.0f), fire_mask[index]);
    v_next[index] = __hfma(__hsub(v_reset, h), spike[index], h);
    fire_mask_next[index] = __float2half(1.0f);
  }
  else
  {
    spike[index] = __float2half(0.0f);
    v_next[index] = h;
  }
}
}

std::vector<at::Tensor> OneSpikeIF_hard_reset_forward(torch::Tensor & x, torch::Tensor & v, torch::Tensor & fire_mask, const float & v_th, const float & v_reset)
{   
    CHECK_TENSOR(x);
    CHECK_TENSOR(v);
    CHECK_TENSOR(fire_mask);
    auto spike = torch::zeros_like(v.data());
    auto v_next = torch::zeros_like(v.data());
    auto fire_mask_next = fire_mask.data().clone();
    CHECK_TENSOR(spike);
    CHECK_TENSOR(v_next);
    CHECK_TENSOR(fire_mask_next);
    const int size = x.numel();
    const int threads = THREADS;
    const int blocks = (size + threads - 1) / threads;
    CHECK_CUDA_OPERATION(hipSetDevice(x.get_device()));
    if (x.scalar_type() == c10::ScalarType::Float)
    {
      OneSpikeIF_hard_reset_forward_cuda_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), v.data_ptr<float>(), fire_mask.data_ptr<float>(), spike.data_ptr<float>(), v_next.data_ptr<float>(), fire_mask_next.data_ptr<float>(),
        v_th, v_reset, size);
    }
    else if (x.scalar_type() == c10::ScalarType::Half)
    {
      OneSpikeIF_hard_reset_forward_cuda_kernel_half<<<blocks, threads>>>(
        x.data_ptr<at::Half>(), v.data_ptr<at::Half>(), fire_mask.data_ptr<at::Half>(), spike.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), fire_mask_next.data_ptr<at::Half>(),
        __float2half(v_th), __float2half(v_reset), size);
    }

    return {spike, v_next, fire_mask_next};
}

__global__ void OneSpikeIF_hard_reset_forward_with_grad_cuda_kernel(
const float* __restrict__ x, const float* __restrict__ v, const float* __restrict__ fire_mask, float* __restrict__ spike, float* __restrict__ v_next, float* __restrict__ fire_mask_next,
float* __restrict__ grad_s_to_h, float* __restrict__ grad_v_to_h, float* __restrict__ grad_s_to_m_last, float* __restrict__ grad_v_to_m_last,
const float v_th, const float v_reset, const int size,
const float alpha, const bool detach_reset, const int grad_surrogate_function_index)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const float h = v[index] + x[index];
  if (h >= v_th)
  {
    spike[index] = 1.0f - fire_mask[index];
    v_next[index] = h + (v_reset - h) * spike[index];
    // v_next[index] = v_reset * spike[index] + (1.0f - spike[index]) * h;
    fire_mask_next[index] = 1.0f;
    grad_s_to_m_last[index] = -1.0f;
  }
  else
  {
    spike[index] = 0.0f;
    v_next[index] = h;
    grad_s_to_m_last[index] = 0.0f;
  }
  grad_s_to_h[index] = (1.0f - fire_mask[index]) * grad_surrogate_function_pointer[grad_surrogate_function_index](alpha, h - v_th);
  const float grad_v_to_s = (v_reset - h) * (1.0f - (float) detach_reset);
  grad_v_to_h[index] = 1.0f - spike[index] + grad_s_to_h[index] * grad_v_to_s;
  grad_v_to_m_last[index] = grad_v_to_s * grad_s_to_m_last[index];

}
}

__global__ void OneSpikeIF_hard_reset_forward_with_grad_cuda_kernel_half(
  const at::Half* __restrict__ x, const at::Half* __restrict__ v, const at::Half* __restrict__ fire_mask, at::Half* __restrict__ spike, at::Half* __restrict__ v_next, at::Half* __restrict__ fire_mask_next,
  at::Half* __restrict__ grad_s_to_h, at::Half* __restrict__ grad_v_to_h, at::Half* __restrict__ grad_s_to_m_last, at::Half* __restrict__ grad_v_to_m_last,
  const half v_th, const half v_reset, const int size,
  const half alpha, const bool detach_reset, const int grad_surrogate_function_index)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const half h = __hadd((half) v[index], (half) x[index]);
    if (__hgeu(h, v_th))
    {
      spike[index] = __hsub(__float2half(1.0f), fire_mask[index]);
      v_next[index] = __hfma(__hsub(v_reset, h), spike[index], h);
      fire_mask_next[index] = __float2half(1.0f);
      grad_s_to_m_last[index] = __float2half(-1.0f);
    }
    else
    {
      spike[index] = __float2half(0.0f);
      v_next[index] = h;
      grad_s_to_m_last[index] = __float2half(0.0f);
    }
    
    grad_s_to_h[index] = __hmul(grad_surrogate_function_pointer_half[grad_surrogate_function_index](alpha, __hsub(h, v_th)), __hsub(__float2half(1.0f), fire_mask[index]));
    const half grad_v_to_s = __hmul(__hsub(v_reset, h), __float2half(1.0f - (float) detach_reset));
    grad_v_to_h[index] = __hfma(grad_s_to_h[index], grad_v_to_s, __hsub(__float2half(1.0f), spike[index]));
    grad_v_to_m_last[index] = __hmul(grad_v_to_s, grad_s_to_m_last[index]);
  }
}

std::vector<at::Tensor> OneSpikeIF_hard_reset_forward_with_grad(torch::Tensor & x, torch::Tensor & v, torch::Tensor & fire_mask, const float & v_th, const float & v_reset,
  const float & alpha, const bool & detach_reset, const int & grad_surrogate_function_index)
{   
  CHECK_TENSOR(x);
  CHECK_TENSOR(v);
  CHECK_TENSOR(fire_mask);

  auto spike = torch::zeros_like(v.data());
  auto v_next = spike.data().clone();
  auto fire_mask_next = fire_mask.data().clone();
  auto grad_s_to_h = spike.data().clone();
  auto grad_v_to_h = spike.data().clone();
  auto grad_s_to_m_last = spike.data().clone();
  auto grad_v_to_m_last = spike.data().clone();

  CHECK_TENSOR(spike);
  CHECK_TENSOR(v_next);
  CHECK_TENSOR(fire_mask_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  CHECK_TENSOR(grad_s_to_m_last);
  CHECK_TENSOR(grad_v_to_m_last);
  const int size = x.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(x.get_device()));
  if (x.scalar_type() == c10::ScalarType::Float)
  {
    OneSpikeIF_hard_reset_forward_with_grad_cuda_kernel<<<blocks, threads>>>(
      x.data_ptr<float>(), v.data_ptr<float>(), fire_mask.data_ptr<float>(), spike.data_ptr<float>(), v_next.data_ptr<float>(), fire_mask_next.data_ptr<float>(),
      grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(), grad_s_to_m_last.data_ptr<float>(), grad_v_to_m_last.data_ptr<float>(),
      v_th, v_reset, size, 
      alpha, detach_reset, grad_surrogate_function_index);
  }
  else if (x.scalar_type() == c10::ScalarType::Half)
  {
    OneSpikeIF_hard_reset_forward_with_grad_cuda_kernel_half<<<blocks, threads>>>(
      x.data_ptr<at::Half>(), v.data_ptr<at::Half>(), fire_mask.data_ptr<at::Half>(), spike.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), fire_mask_next.data_ptr<at::Half>(),
      grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(), grad_s_to_m_last.data_ptr<at::Half>(), grad_v_to_m_last.data_ptr<at::Half>(),
      __float2half(v_th), __float2half(v_reset), size, 
      __float2half(alpha), detach_reset, grad_surrogate_function_index);
  }

  return {spike, v_next, fire_mask_next, grad_s_to_h, grad_v_to_h, grad_s_to_m_last, grad_v_to_m_last};
}



//OneSpikeIF hard reset fptt----------------------------------------------------
__global__ void OneSpikeIF_hard_reset_fptt_cuda_kernel(
  const float* __restrict__ x_seq, float* __restrict__ spike_seq, float* __restrict__ v_next, float* __restrict__ fire_mask_next, 
  const float v_th, const float v_reset, const int neuron_num, const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const float h = v_next[index] + x_seq[mem_index];
      if (h >= v_th)
      {
        spike_seq[mem_index] = 1.0f - fire_mask_next[index];
        v_next[index] = h + (v_reset - h) * spike_seq[mem_index];
        fire_mask_next[index] = 1.0f;
      }
      else
      {
        spike_seq[mem_index] = 0.0f;
        v_next[index] = h;
      }
    }
    
  }
}

__global__ void OneSpikeIF_hard_reset_fptt_cuda_kernel_half(
  const at::Half* __restrict__ x_seq, at::Half* __restrict__ spike_seq, at::Half* __restrict__ v_next, at::Half* __restrict__ fire_mask_next, 
  const half v_th, const half v_reset, const int neuron_num, const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const half h = __hadd((half) v_next[index], (half) x_seq[mem_index]);
      if (__hgeu(h, v_th))
      {
        spike_seq[mem_index] = __hsub(__float2half(1.0f), fire_mask_next[index]);
        v_next[index] = __hfma(__hsub(v_reset, h), spike_seq[mem_index], h);
        fire_mask_next[index] = __float2half(1.0f);
      }
      else
      {
        spike_seq[mem_index] = __float2half(0.0f);
        v_next[index] = h;
      }
    }
    
  }
}

std::vector<at::Tensor> OneSpikeIF_hard_reset_fptt(torch::Tensor & x_seq, torch::Tensor & v, torch::Tensor & fire_mask, const float & v_th, const float & v_reset)
{
    CHECK_TENSOR(x_seq);
    CHECK_TENSOR(v);
    CHECK_TENSOR(fire_mask);
    auto spike_seq = torch::zeros_like(x_seq.data());
    auto v_next = v.data().clone();
    auto fire_mask_next = fire_mask.data().clone();
    CHECK_TENSOR(spike_seq);
    CHECK_TENSOR(v_next);
    CHECK_TENSOR(fire_mask_next);
    const int seq_len = x_seq.size(0);
    const int size = x_seq.numel();
    const int threads = THREADS;
    const int neuron_num = size / seq_len;
    const int blocks = (neuron_num + threads - 1) / threads;
    CHECK_CUDA_OPERATION(hipSetDevice(x_seq.get_device()));
    if (x_seq.scalar_type() == c10::ScalarType::Float)
    {
      OneSpikeIF_hard_reset_fptt_cuda_kernel<<<blocks, threads>>>(
        x_seq.data_ptr<float>(), spike_seq.data_ptr<float>(), v_next.data_ptr<float>(), fire_mask_next.data_ptr<float>(),
        v_th, v_reset, neuron_num, size);
    }
    else if (x_seq.scalar_type() == c10::ScalarType::Half)
    {
      OneSpikeIF_hard_reset_fptt_cuda_kernel_half<<<blocks, threads>>>(
        x_seq.data_ptr<at::Half>(), spike_seq.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), fire_mask_next.data_ptr<at::Half>(),
        __float2half(v_th), __float2half(v_reset), neuron_num, size);
    }

    return {spike_seq, v_next, fire_mask_next};
}

__global__ void OneSpikeIF_hard_reset_fptt_with_grad_cuda_kernel(
  const float* __restrict__ x_seq, float* __restrict__ spike_seq, float* __restrict__ v_next, float* __restrict__ fire_mask_next, float* __restrict__ grad_s_to_h, float* __restrict__ grad_v_to_h, float* __restrict__ grad_s_to_m_last, float* __restrict__ grad_v_to_m_last, 
  const float v_th, const float v_reset, const int neuron_num, const int size,
  const float alpha, const bool detach_reset, const int grad_surrogate_function_index)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const float h = v_next[index] + x_seq[mem_index];
      const float fire_mask_last = fire_mask_next[index];
      if (h >= v_th)
      {
        spike_seq[mem_index] = 1.0f - fire_mask_next[index];
        v_next[index] = h + (v_reset - h) * spike_seq[mem_index];
        fire_mask_next[index] = 1.0f;
        grad_s_to_m_last[mem_index] = -1.0f;
      }
      else
      {
        spike_seq[mem_index] = 0.0f;
        v_next[index] = h;
        grad_s_to_m_last[mem_index] = 0.0f;
      }
      grad_s_to_h[mem_index] = (1.0f - fire_mask_last) * grad_surrogate_function_pointer[grad_surrogate_function_index](alpha, h - v_th);
      const float grad_v_to_s = (v_reset - h) * (1.0f - (float) detach_reset);
      grad_v_to_h[mem_index] = 1.0f - spike_seq[mem_index] + grad_s_to_h[mem_index] * grad_v_to_s;
      grad_v_to_m_last[mem_index] = grad_v_to_s * grad_s_to_m_last[mem_index];
    }
    
  }
}

__global__ void OneSpikeIF_hard_reset_fptt_with_grad_cuda_kernel_half(
  const at::Half* __restrict__ x_seq, at::Half* __restrict__ spike_seq, at::Half* __restrict__ v_next, at::Half* __restrict__ fire_mask_next, at::Half* __restrict__ grad_s_to_h, at::Half* __restrict__ grad_v_to_h, at::Half* __restrict__ grad_s_to_m_last, at::Half* __restrict__ grad_v_to_m_last,
  const half v_th, const half v_reset, const int neuron_num, const int size,
  const half alpha, const bool detach_reset, const int grad_surrogate_function_index)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = 0; mem_offset < size; mem_offset += neuron_num)
    {
      const int mem_index = index + mem_offset;
      const half h = __hadd((half) v_next[index], (half) x_seq[mem_index]);
      const half fire_mask_last = fire_mask_next[index];
      if (__hgeu(h, v_th))
      {
        spike_seq[mem_index] = __hsub(__float2half(1.0f), fire_mask_next[index]);
        v_next[index] = __hfma(__hsub(v_reset, h), spike_seq[mem_index], h);
        fire_mask_next[index] = __float2half(1.0f);
        grad_s_to_m_last[mem_index] = __float2half(-1.0f);
      }
      else
      {
        spike_seq[mem_index] = __float2half(0.0f);
        v_next[index] = h;
        grad_s_to_m_last[mem_index] = __float2half(0.0f);
      }
      grad_s_to_h[mem_index] = __hmul(grad_surrogate_function_pointer_half[grad_surrogate_function_index](alpha, __hsub(h, v_th)), __hsub(__float2half(1.0f), fire_mask_last));
      const half grad_v_to_s = __hmul(__hsub(v_reset, h), __float2half(1.0f - (float) detach_reset));
      grad_v_to_h[mem_index] = __hfma(grad_s_to_h[mem_index], grad_v_to_s, __hsub(__float2half(1.0f), spike_seq[mem_index]));
      grad_v_to_m_last[index] = __hmul(grad_v_to_s, grad_s_to_m_last[mem_index]);
    }
    
  }
}

std::vector<at::Tensor> OneSpikeIF_hard_reset_fptt_with_grad(torch::Tensor & x_seq, torch::Tensor & v, torch::Tensor & fire_mask, const float & v_th, const float & v_reset, 
  const float & alpha, const bool & detach_reset, const int & grad_surrogate_function_index)
{
  CHECK_TENSOR(x_seq);
  CHECK_TENSOR(v);
  CHECK_TENSOR(fire_mask);
  auto spike_seq = torch::zeros_like(x_seq.data());
  auto v_next = v.data().clone();
  auto fire_mask_next = fire_mask.data().clone();
  auto grad_s_to_h = spike_seq.data().clone();
  auto grad_v_to_h = spike_seq.data().clone();
  auto grad_s_to_m_last = spike_seq.data().clone();
  auto grad_v_to_m_last = spike_seq.data().clone();
  CHECK_TENSOR(spike_seq);
  CHECK_TENSOR(v_next);
  CHECK_TENSOR(fire_mask_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  CHECK_TENSOR(grad_s_to_m_last);
  CHECK_TENSOR(grad_v_to_m_last);
  const int seq_len = x_seq.size(0);
  const int size = x_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(x_seq.get_device()));
  if (x_seq.scalar_type() == c10::ScalarType::Float)
  {
    OneSpikeIF_hard_reset_fptt_with_grad_cuda_kernel<<<blocks, threads>>>(
      x_seq.data_ptr<float>(), spike_seq.data_ptr<float>(), v_next.data_ptr<float>(), fire_mask_next.data_ptr<float>(), 
      grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(), grad_s_to_m_last.data_ptr<float>(), grad_v_to_m_last.data_ptr<float>(),
      v_th, v_reset, neuron_num, size, 
      alpha, detach_reset, grad_surrogate_function_index);
  }
  else if (x_seq.scalar_type() == c10::ScalarType::Half)
  {
    OneSpikeIF_hard_reset_fptt_with_grad_cuda_kernel_half<<<blocks, threads>>>(
      x_seq.data_ptr<at::Half>(), spike_seq.data_ptr<at::Half>(), v_next.data_ptr<at::Half>(), fire_mask_next.data_ptr<at::Half>(), 
      grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(), grad_s_to_m_last.data_ptr<at::Half>(), grad_v_to_m_last.data_ptr<at::Half>(),
      __float2half(v_th), __float2half(v_reset), neuron_num, size, 
      __float2half(alpha), detach_reset, grad_surrogate_function_index);
  }

  return {spike_seq, v_next, fire_mask_next, grad_s_to_h, grad_v_to_h, grad_s_to_m_last, grad_v_to_m_last};
}