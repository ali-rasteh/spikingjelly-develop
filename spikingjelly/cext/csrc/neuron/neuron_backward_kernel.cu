#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <math.h>
#include <stdio.h>
#include <torch/extension.h>
#include "neuron_def.h"

//LIF bp----------------------------------------------------
__global__ void LIF_backward_cuda_kernel(
    float* __restrict__ grad_x, float* __restrict__ grad_v,
    const float* __restrict__ grad_spike, const float* __restrict__ grad_v_next, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
    const int size,
    const float reciprocal_tau, const float one_sub_reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const float grad_h = grad_spike[index] * grad_s_to_h[index] + grad_v_next[index] * grad_v_to_h[index];
    grad_x[index] = grad_h * reciprocal_tau;
    grad_v[index] = grad_h * one_sub_reciprocal_tau;
  }
}

__global__ void LIF_backward_cuda_kernel_half(
  c10::Half* __restrict__ grad_x, c10::Half* __restrict__ grad_v,
  const c10::Half* __restrict__ grad_spike, const c10::Half* __restrict__ grad_v_next, const c10::Half* __restrict__ grad_s_to_h, const c10::Half* __restrict__ grad_v_to_h,
  const int size,
  const half reciprocal_tau, const half one_sub_reciprocal_tau)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const half grad_h = __hfma(grad_spike[index], grad_s_to_h[index], __hmul(grad_v_next[index], grad_v_to_h[index]));
  grad_x[index] = __hmul(grad_h, reciprocal_tau);
  grad_v[index] = __hmul(grad_h, one_sub_reciprocal_tau);
}
}
//detach x--------------
__global__ void LIF_detach_x_backward_cuda_kernel(
  float* __restrict__ grad_x, float* __restrict__ grad_v,
  const float* __restrict__ grad_spike, const float* __restrict__ grad_v_next, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
  const int size,
  const float one_sub_reciprocal_tau)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
  const float grad_h = grad_spike[index] * grad_s_to_h[index] + grad_v_next[index] * grad_v_to_h[index];
  grad_x[index] = grad_h;
  grad_v[index] = grad_h * one_sub_reciprocal_tau;
}
}

__global__ void LIF_detach_x_backward_cuda_kernel_half(
c10::Half* __restrict__ grad_x, c10::Half* __restrict__ grad_v,
const c10::Half* __restrict__ grad_spike, const c10::Half* __restrict__ grad_v_next, const c10::Half* __restrict__ grad_s_to_h, const c10::Half* __restrict__ grad_v_to_h,
const int size,
const half one_sub_reciprocal_tau)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
{
const half grad_h = __hfma(grad_spike[index], grad_s_to_h[index], __hmul(grad_v_next[index], grad_v_to_h[index]));
grad_x[index] = grad_h;
grad_v[index] = __hmul(grad_h, one_sub_reciprocal_tau);
}
}

std::vector<at::Tensor> LIF_backward(
  torch::Tensor & grad_spike, torch::Tensor & grad_v_next, torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h,
  const float & reciprocal_tau, const bool & detach_x)
{
  CHECK_TENSOR(grad_spike);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  auto grad_x = torch::zeros_like(grad_spike.data());
  auto grad_v = grad_x.data().clone();
  CHECK_TENSOR(grad_x);
  CHECK_TENSOR(grad_v);
  const int size = grad_spike.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike.get_device()));
  if (grad_x.scalar_type() == c10::ScalarType::Float)
  {
    if (detach_x)
    {
      LIF_detach_x_backward_cuda_kernel<<<blocks, threads>>>(
        grad_x.data_ptr<float>(), grad_v.data_ptr<float>(),
        grad_spike.data_ptr<float>(), grad_v_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
        size, 1.0f - reciprocal_tau);
    }
    else
    {
      LIF_backward_cuda_kernel<<<blocks, threads>>>(
        grad_x.data_ptr<float>(), grad_v.data_ptr<float>(),
        grad_spike.data_ptr<float>(), grad_v_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
        size, reciprocal_tau, 1.0f - reciprocal_tau);
    }

  }
  else if (grad_x.scalar_type() == c10::ScalarType::Half)
  {
    if (detach_x)
    {
      LIF_detach_x_backward_cuda_kernel_half<<<blocks, threads>>>(
        grad_x.data_ptr<at::Half>(), grad_v.data_ptr<at::Half>(),
        grad_spike.data_ptr<at::Half>(), grad_v_next.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
        size, __float2half(1.0f - reciprocal_tau));
    }
    else
    {
      LIF_backward_cuda_kernel_half<<<blocks, threads>>>(
        grad_x.data_ptr<at::Half>(), grad_v.data_ptr<at::Half>(),
        grad_spike.data_ptr<at::Half>(), grad_v_next.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
        size, __float2half(reciprocal_tau), __float2half(1.0f - reciprocal_tau));
    }

  }
  
  return {grad_x, grad_v};
}

//LIF bptt----------------------------------------------------

__global__ void LIF_bptt_cuda_kernel(
  float* __restrict__ grad_x_seq, float* __restrict__ grad_v,
  const float* __restrict__ grad_spike_seq, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
  const int neuron_num, const int size,
  const float reciprocal_tau, const float one_sub_reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    float grad_h;
    for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
    {
      const int mem_index = index + mem_offset;
      grad_h = grad_spike_seq[mem_index] * grad_s_to_h[mem_index] + grad_v[index] * grad_v_to_h[mem_index];
      grad_x_seq[mem_index] = grad_h * reciprocal_tau;
      grad_v[index] = grad_h * one_sub_reciprocal_tau;
    }
  }
}

__global__ void LIF_bptt_cuda_kernel_half(
  at::Half* __restrict__ grad_x_seq, at::Half* __restrict__ grad_v,
  const at::Half* __restrict__ grad_spike_seq, const at::Half* __restrict__ grad_s_to_h, const at::Half* __restrict__ grad_v_to_h,
  const int neuron_num, const int size,
  const half reciprocal_tau, const half one_sub_reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    half grad_h;
    for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
    {
      const int mem_index = index + mem_offset;
      grad_h = __hfma(grad_spike_seq[mem_index], grad_s_to_h[mem_index], __hmul(grad_v[index], grad_v_to_h[mem_index]));
      grad_x_seq[mem_index] = __hmul(grad_h, reciprocal_tau);
      grad_v[index] = __hmul(grad_h, one_sub_reciprocal_tau);
    }
  }
}

//detach x------

__global__ void LIF_detach_x_bptt_cuda_kernel(
  float* __restrict__ grad_x_seq, float* __restrict__ grad_v,
  const float* __restrict__ grad_spike_seq, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
  const int neuron_num, const int size,
  const float one_sub_reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    float grad_h;
    for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
    {
      const int mem_index = index + mem_offset;
      grad_h = grad_spike_seq[mem_index] * grad_s_to_h[mem_index] + grad_v[index] * grad_v_to_h[mem_index];
      grad_x_seq[mem_index] = grad_h;
      grad_v[index] = grad_h * one_sub_reciprocal_tau;
    }
  }
}

__global__ void LIF_detach_x_bptt_cuda_kernel_half(
  at::Half* __restrict__ grad_x_seq, at::Half* __restrict__ grad_v,
  const at::Half* __restrict__ grad_spike_seq, const at::Half* __restrict__ grad_s_to_h, const at::Half* __restrict__ grad_v_to_h,
  const int neuron_num, const int size,
  const half one_sub_reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    half grad_h;
    for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
    {
      const int mem_index = index + mem_offset;
      grad_h = __hfma(grad_spike_seq[mem_index], grad_s_to_h[mem_index], __hmul(grad_v[index], grad_v_to_h[mem_index]));
      grad_x_seq[mem_index] = grad_h;
      grad_v[index] = __hmul(grad_h, one_sub_reciprocal_tau);
    }
  }
}

std::vector<at::Tensor> LIF_bptt(
  torch::Tensor & grad_spike_seq, torch::Tensor & grad_v_next,
  torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h,
  const float & reciprocal_tau, const bool & detach_x)
{
  CHECK_TENSOR(grad_spike_seq);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  auto grad_x_seq = torch::zeros_like(grad_spike_seq.data());
  auto grad_v = grad_v_next.data().clone();
  CHECK_TENSOR(grad_x_seq);
  CHECK_TENSOR(grad_v);
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike_seq.get_device()));
  const int seq_len = grad_spike_seq.size(0);
  const int size = grad_spike_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  if (grad_x_seq.scalar_type() == c10::ScalarType::Float)
  {
    if (detach_x)
    {
      LIF_detach_x_bptt_cuda_kernel<<<blocks, threads>>>(
        grad_x_seq.data_ptr<float>(), grad_v.data_ptr<float>(),
        grad_spike_seq.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
        neuron_num, size,
        1.0f - reciprocal_tau);
    }
    else
    {
      LIF_bptt_cuda_kernel<<<blocks, threads>>>(
        grad_x_seq.data_ptr<float>(), grad_v.data_ptr<float>(),
        grad_spike_seq.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
        neuron_num, size,
        reciprocal_tau, 1.0f - reciprocal_tau);
    }

  }
  else if (grad_x_seq.scalar_type() == c10::ScalarType::Half)
  {
    if (detach_x)
    {
      LIF_detach_x_bptt_cuda_kernel_half<<<blocks, threads>>>(
        grad_x_seq.data_ptr<at::Half>(), grad_v.data_ptr<at::Half>(),
        grad_spike_seq.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
        neuron_num, size,
        __float2half(1.0f - reciprocal_tau));
    }
    else
    {
      LIF_bptt_cuda_kernel_half<<<blocks, threads>>>(
        grad_x_seq.data_ptr<at::Half>(), grad_v.data_ptr<at::Half>(),
        grad_spike_seq.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
        neuron_num, size,
        __float2half(reciprocal_tau), __float2half(1.0f - reciprocal_tau));
    }

  }
  return {grad_x_seq, grad_v};
}

//IF bp----------------------------------------------------
__global__ void IF_backward_cuda_kernel(
  float* __restrict__ grad_x, float* __restrict__ grad_v,
  const float* __restrict__ grad_spike, const float* __restrict__ grad_v_next, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
  const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const float grad_h = grad_spike[index] * grad_s_to_h[index] + grad_v_next[index] * grad_v_to_h[index];
    grad_x[index] = grad_h;
    grad_v[index] = grad_h;
  }
}

__global__ void IF_backward_cuda_kernel_half(
  at::Half* __restrict__ grad_x, at::Half* __restrict__ grad_v,
  const at::Half* __restrict__ grad_spike, const at::Half* __restrict__ grad_v_next, const at::Half* __restrict__ grad_s_to_h, const at::Half* __restrict__ grad_v_to_h,
  const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const half grad_h = __hfma(grad_spike[index], grad_s_to_h[index], __hmul(grad_v_next[index], grad_v_to_h[index]));
    grad_x[index] = grad_h;
    grad_v[index] = grad_h;
  }
}

std::vector<at::Tensor> IF_backward(
  torch::Tensor & grad_spike, torch::Tensor & grad_v_next, torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h)
{
  CHECK_TENSOR(grad_spike);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  auto grad_x = torch::zeros_like(grad_spike.data());
  auto grad_v = grad_x.data().clone();
  CHECK_TENSOR(grad_x);
  CHECK_TENSOR(grad_v);
  const int size = grad_spike.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike.get_device()));
  if (grad_spike.scalar_type() == c10::ScalarType::Float)
  {
    IF_backward_cuda_kernel<<<blocks, threads>>>(
      grad_x.data_ptr<float>(), grad_v.data_ptr<float>(),
      grad_spike.data_ptr<float>(), grad_v_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
      size);
  }
  else if (grad_spike.scalar_type() == c10::ScalarType::Half)
  {
    IF_backward_cuda_kernel_half<<<blocks, threads>>>(
      grad_x.data_ptr<at::Half>(), grad_v.data_ptr<at::Half>(),
      grad_spike.data_ptr<at::Half>(), grad_v_next.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
      size);
  }


  return {grad_x, grad_v};
}

//IF bptt----------------------------------------------------

__global__ void IF_bptt_cuda_kernel(
float* __restrict__ grad_x_seq, float* __restrict__ grad_v,
const float* __restrict__ grad_spike_seq, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
const int neuron_num, const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    float grad_h;
    for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
    {
      const int mem_index = index + mem_offset;
      grad_h = grad_spike_seq[mem_index] * grad_s_to_h[mem_index] + grad_v[index] * grad_v_to_h[mem_index];
      grad_x_seq[mem_index] = grad_h;
      grad_v[index] = grad_h;
    }
  }
}

__global__ void IF_bptt_cuda_kernel_half(
  at::Half* __restrict__ grad_x_seq, at::Half* __restrict__ grad_v,
  const at::Half* __restrict__ grad_spike_seq, const at::Half* __restrict__ grad_s_to_h, const at::Half* __restrict__ grad_v_to_h,
  const int neuron_num, const int size)
  {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < neuron_num)
    {
      half grad_h;
      for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
      {
        const int mem_index = index + mem_offset;
        grad_h = __hfma(grad_spike_seq[mem_index], grad_s_to_h[mem_index], __hmul(grad_v[index], grad_v_to_h[mem_index]));
        grad_x_seq[mem_index] = grad_h;
        grad_v[index] = grad_h;
      }
    }
  }

std::vector<at::Tensor> IF_bptt(
  torch::Tensor & grad_spike_seq, torch::Tensor & grad_v_next,
  torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h)
{
  CHECK_TENSOR(grad_spike_seq);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  auto grad_x_seq = torch::zeros_like(grad_spike_seq.data());
  auto grad_v = grad_v_next.data().clone();
  CHECK_TENSOR(grad_x_seq);
  CHECK_TENSOR(grad_v);
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike_seq.get_device()));
  const int seq_len = grad_spike_seq.size(0);
  const int size = grad_spike_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  if (grad_x_seq.scalar_type() == c10::ScalarType::Float)
  {
    IF_bptt_cuda_kernel<<<blocks, threads>>>(
      grad_x_seq.data_ptr<float>(), grad_v.data_ptr<float>(),
      grad_spike_seq.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
      neuron_num, size);
  }
  else if (grad_x_seq.scalar_type() == c10::ScalarType::Half)
  {
    IF_bptt_cuda_kernel_half<<<blocks, threads>>>(
      grad_x_seq.data_ptr<at::Half>(), grad_v.data_ptr<at::Half>(),
      grad_spike_seq.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(),
      neuron_num, size);
  }

  return {grad_x_seq, grad_v};
}


//OneSpikeIF bp----------------------------------------------------
__global__ void OneSpikeIF_backward_cuda_kernel(
  float* __restrict__ grad_x, float* __restrict__ grad_v, float* __restrict__ grad_m,
  const float* __restrict__ grad_spike, const float* __restrict__ grad_v_next, const float* __restrict__ grad_fire_mask_next, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h, const float* __restrict__ grad_s_to_m_last, const float* __restrict__ grad_v_to_m_last,
  const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const float grad_h = grad_v_next[index] * grad_v_to_h[index] + (grad_spike[index] + grad_fire_mask_next[index]) * grad_s_to_h[index];
    grad_x[index] = grad_h;
    grad_v[index] = grad_h;
    grad_m[index] = (grad_fire_mask_next[index] + grad_spike[index]) * grad_s_to_m_last[index] + grad_v_next[index] * grad_v_to_m_last[index] + grad_fire_mask_next[index];
  }
}

__global__ void OneSpikeIF_backward_cuda_kernel_half(
  at::Half* __restrict__ grad_x, at::Half* __restrict__ grad_v, at::Half* __restrict__ grad_m,
  const at::Half* __restrict__ grad_spike, const at::Half* __restrict__ grad_v_next, const at::Half* __restrict__ grad_fire_mask_next, const at::Half* __restrict__ grad_s_to_h, const at::Half* __restrict__ grad_v_to_h, const at::Half* __restrict__ grad_s_to_m_last, const at::Half* __restrict__ grad_v_to_m_last,
  const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const half grad_h = __hfma(grad_v_next[index], grad_v_to_h[index], __hmul(__hadd((half) grad_fire_mask_next[index], (half) grad_spike[index]), grad_s_to_h[index]));
    grad_x[index] = grad_h;
    grad_v[index] = grad_h;
    grad_m[index] =  __hadd(__hfma(grad_v_next[index], grad_v_to_m_last[index], __hmul(__hadd((half) grad_fire_mask_next[index], (half) grad_spike[index]), grad_s_to_m_last[index])), (half) grad_fire_mask_next[index]);
  }
}


std::vector<at::Tensor> OneSpikeIF_backward(
  torch::Tensor & grad_spike, torch::Tensor & grad_v_next, torch::Tensor & grad_fire_mask_next, torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h, torch::Tensor & grad_s_to_m_last, torch::Tensor & grad_v_to_m_last)
{
  CHECK_TENSOR(grad_spike);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_fire_mask_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  CHECK_TENSOR(grad_s_to_m_last);
  CHECK_TENSOR(grad_v_to_m_last);
  auto grad_x = torch::zeros_like(grad_spike.data());
  auto grad_v = grad_x.data().clone();
  auto grad_m = grad_x.data().clone();
  CHECK_TENSOR(grad_x);
  CHECK_TENSOR(grad_v);
  CHECK_TENSOR(grad_m);
  const int size = grad_spike.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike.get_device()));
  if (grad_spike.scalar_type() == c10::ScalarType::Float)
  {
    OneSpikeIF_backward_cuda_kernel<<<blocks, threads>>>(
      grad_x.data_ptr<float>(), grad_v.data_ptr<float>(), grad_m.data_ptr<float>(),
      grad_spike.data_ptr<float>(), grad_v_next.data_ptr<float>(), grad_fire_mask_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(), grad_s_to_m_last.data_ptr<float>(), grad_v_to_m_last.data_ptr<float>(),
      size);
  }
  else if (grad_spike.scalar_type() == c10::ScalarType::Half)
  {
    OneSpikeIF_backward_cuda_kernel_half<<<blocks, threads>>>(
      grad_x.data_ptr<at::Half>(), grad_v.data_ptr<at::Half>(), grad_m.data_ptr<at::Half>(),
      grad_spike.data_ptr<at::Half>(), grad_v_next.data_ptr<at::Half>(), grad_fire_mask_next.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(), grad_s_to_m_last.data_ptr<at::Half>(), grad_v_to_m_last.data_ptr<at::Half>(),
      size);
  }


  return {grad_x, grad_v, grad_m};
}

//OneSpikeIF bptt----------------------------------------------------

__global__ void OneSpikeIF_bptt_cuda_kernel(
float* __restrict__ grad_x_seq, float* __restrict__ grad_v, float* __restrict__ grad_m,
const float* __restrict__ grad_spike_seq, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h, const float* __restrict__ grad_s_to_m_last, const float* __restrict__ grad_v_to_m_last,
const int neuron_num, const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
    {
      const int mem_index = index + mem_offset;

      const float grad_h = grad_v[index] * grad_v_to_h[mem_index] + (grad_spike_seq[mem_index] + grad_m[index]) * grad_s_to_h[mem_index];
      grad_m[index] += (grad_m[index] + grad_spike_seq[mem_index]) * grad_s_to_m_last[mem_index] + grad_v[index] * grad_v_to_m_last[mem_index];
      grad_x_seq[mem_index] = grad_h;
      grad_v[index] = grad_h;
    }
  }
}

__global__ void OneSpikeIF_bptt_cuda_kernel_half(
  at::Half* __restrict__ grad_x_seq, at::Half* __restrict__ grad_v, at::Half* __restrict__ grad_m,
  const at::Half* __restrict__ grad_spike_seq, const at::Half* __restrict__ grad_s_to_h, const at::Half* __restrict__ grad_v_to_h, const at::Half* __restrict__ grad_s_to_m_last, const at::Half* __restrict__ grad_v_to_m_last,
  const int neuron_num, const int size)
  {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < neuron_num)
    {
      for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
      {
        const int mem_index = index + mem_offset;
        const half grad_h = __hfma(grad_v[index], grad_v_to_h[mem_index], __hmul(__hadd((half) grad_m[index], (half) grad_spike_seq[mem_index]), grad_s_to_h[mem_index]));
        grad_m[index] = __hadd((half) grad_m[index], __hfma(grad_v[index], grad_v_to_m_last[mem_index], __hmul(__hadd((half) grad_m[index], (half) grad_spike_seq[mem_index]), grad_s_to_m_last[mem_index])));
        grad_x_seq[mem_index] = grad_h;
        grad_v[index] = grad_h;
        
      }
    }
  }

std::vector<at::Tensor> OneSpikeIF_bptt(
  torch::Tensor & grad_spike_seq, torch::Tensor & grad_v_next, torch::Tensor & grad_fire_mask_next,
  torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h, torch::Tensor & grad_s_to_m_last, torch::Tensor & grad_v_to_m_last)
{
  CHECK_TENSOR(grad_spike_seq);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_fire_mask_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  CHECK_TENSOR(grad_s_to_m_last);
  CHECK_TENSOR(grad_v_to_m_last);
  auto grad_x_seq = torch::zeros_like(grad_spike_seq.data());
  auto grad_v = grad_v_next.data().clone();
  auto grad_m = grad_fire_mask_next.data().clone();
  CHECK_TENSOR(grad_x_seq);
  CHECK_TENSOR(grad_v);
  CHECK_TENSOR(grad_m);
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike_seq.get_device()));
  const int seq_len = grad_spike_seq.size(0);
  const int size = grad_spike_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  if (grad_x_seq.scalar_type() == c10::ScalarType::Float)
  {
    OneSpikeIF_bptt_cuda_kernel<<<blocks, threads>>>(
      grad_x_seq.data_ptr<float>(), grad_v.data_ptr<float>(), grad_m.data_ptr<float>(),
      grad_spike_seq.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(), grad_s_to_m_last.data_ptr<float>(), grad_v_to_m_last.data_ptr<float>(),
      neuron_num, size);
  }
  else if (grad_x_seq.scalar_type() == c10::ScalarType::Half)
  {
    OneSpikeIF_bptt_cuda_kernel_half<<<blocks, threads>>>(
      grad_x_seq.data_ptr<at::Half>(), grad_v.data_ptr<at::Half>(), grad_m.data_ptr<at::Half>(),
      grad_spike_seq.data_ptr<at::Half>(), grad_s_to_h.data_ptr<at::Half>(), grad_v_to_h.data_ptr<at::Half>(), grad_s_to_m_last.data_ptr<at::Half>(), grad_v_to_m_last.data_ptr<at::Half>(),
      neuron_num, size);
  }

  return {grad_x_seq, grad_v, grad_m};
}
